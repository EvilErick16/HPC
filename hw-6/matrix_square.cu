/* 
Erick Juarez
CPSC 479 Sec 1
HOMEWORK 6 - 4/20/20
tested using nvcc - CUDA compiler driver release 9.0, V9.0.176
*/

#include <stdio.h>
#include <hip/hip_runtime.h>

// device function: perform calculations to square a matrix 
__global__ void square(int *matrix, int *result, int matrix_size){
    // solves 1 row per thread 
    int row_id = threadIdx.x; 
    for (int col_ix = 0; col_ix < matrix_size; col_ix++){
        for (int row_ix = 0; row_ix < matrix_size; row_ix++){
            result[row_id * matrix_size + col_ix] += matrix[row_id * matrix_size + row_ix] * matrix[row_ix * matrix_size + col_ix];
        }
    }
}

// device function is used to initalize both matrices in parallel 
__global__ void init(int * matrix, int * result){
    result[threadIdx.x] = 0; 
    matrix[threadIdx.x] = threadIdx.x +1; 
}

// Main program: initialization 
int main(int argc, char* argv[]){  
    const int RLEN = 32;            // matrix of size RLEN x RLEN (matrix has to be a sqaure) 
    const int MSIZE = RLEN * RLEN;  // total number of elements in the matrix 

    // Allocate memory on host for inpaut and output 
    int *h_matrix, *h_result;

    // allocate memory on device for copy of input and output 
    int *d_matrix, *d_result;
    int byte_size = MSIZE * sizeof(int);
    dim3 b_dim(MSIZE);
    hipMalloc((void**) &d_matrix, byte_size);
    hipMalloc((void**) &d_result, byte_size);

    // initialize device matrix and result matrix. Then square original matrix and place result in result matrix
    init<<<1, b_dim>>>(d_matrix, d_result);
    square<<<1, b_dim>>>(d_matrix, d_result, RLEN);

    // copy device ouput to host and cleanup 
    h_matrix = (int *) malloc(byte_size);
    h_result = (int *) malloc(byte_size);
    hipMemcpy(h_matrix, d_matrix, byte_size, hipMemcpyDeviceToHost);
    hipMemcpy(h_result, d_result, byte_size, hipMemcpyDeviceToHost);
    hipFree(d_matrix);
    hipFree(d_result);

    // Print results
    printf("==========Original Matrix==========\n");
    for (int i = 0; i < RLEN; i++){
        for (int k = 0; k < RLEN; k++){
            printf("[%d] ", h_matrix[RLEN * i + k]);
        }
        printf("\n");
    }
    printf("==========Squared Matrix==========\n");
    for (int i = 0; i < RLEN; i++){
        for (int k = 0; k < RLEN; k++){
            printf("[%d] ", h_result[RLEN * i + k]);
        }
        printf("\n");
    }
    free(h_matrix);
    free(h_result);
    return 0; 
}