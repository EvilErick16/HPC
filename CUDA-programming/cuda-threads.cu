/* 
Erick Juarez
CPSC 479 Sec 1
HOMEWORK 5 - 3/25/20
tested using nvcc - CUDA compiler driver release 9.0, V9.0.176
*/

#include <stdio.h>
#include <hip/hip_runtime.h>
#define P1 16     // Size of array for problem 1
#define P2 1024   // Size of array for problems 2 & 3
#define P4 8000   // Size of array for problem 4


// device function to initialize array - used in problems 1 & 2 
__global__ void initialize(int *array){
    array[threadIdx.x] = 0;
}

// device function will add values to array - used in problems 3 & 4
__global__ void add(int *array){
  array[threadIdx.x] += threadIdx.x;
}

// Main function 
int main(int argc, char * argv[]) {

    int *host_array;     // host copy of array 
    int *dev_array;      // device copy of array   
    int byte_size;       // size in bytes of an object 

    /////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Problem 1 - initialize array of size 16 to all zeroes 
    byte_size = P1 * sizeof(int);                 // get size of 16 integers in bytes 
    dim3 blockd(P1);                            // Create dim3 type with value of 16 on the first dimension   
    hipMalloc((void **) &dev_array, byte_size);  // Allocate memory on the device 
    initialize<<<1, blockd>>>(dev_array);       // Launch new kernel on device with 16 threads 
    host_array = (int *)malloc(byte_size);        // Allocate memory for the host
    hipMemcpy(host_array, dev_array, byte_size, hipMemcpyDeviceToHost); // Copy from device to host 

    // Verify that array was allocated and copied properly to the host, then cleanup 
    try {
      for(int c = 0; c < P1; c++){
        if(host_array[c] != 0) {throw "NON_ZERO_ELEM";}
      }
    } catch (...) {
      printf("Problem 1 - Elements not initialized properly!\n");
      hipFree(dev_array);
      free(host_array);
      return 0; 
    }
    printf("Problem 1 - Successfully initialized array with %d elements\n", P1); 
    hipFree(dev_array);
    free(host_array); 

    /////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Problem 2 - initialize array of size 1024 to all zeroes
    byte_size = P2 * sizeof(int);                // get size of 1024 integers in bytes
    dim3 bd(P2);                                 // create dim3 object with 1024 as the first dimensoin 
    hipMalloc((void**) &dev_array, byte_size);  // Allocate memory on device 
    initialize<<<1, bd>>>(dev_array);            // Launch kernel on device with 1024 threads 
    host_array = (int*)malloc(byte_size);        // Allocate memory on the host 
    hipMemcpy(host_array, dev_array, byte_size, hipMemcpyDeviceToHost); // Copy from device to host 

    // Verify that array was allocated and copied properly, then cleanup
    try {
      for(int c = 0; c < P2; c++){
        if(host_array[c] != 0) {throw "NON_ZERO_ELEM";}
      }
    } catch (...) {
      printf("Problem 2 - Elements not initialized properly!\n");
      hipFree(dev_array);
      free(host_array);
      return 0; 
    }
    printf("Problem 2 - Successfully initialized array with %d elements\n", P2);
   
    /////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Problem 3 - add i to array[i] 
    add<<<1, bd>>>(dev_array);          // Launch add kernel with array from the previous problem 
    hipMemcpy(host_array, dev_array, byte_size, hipMemcpyDeviceToHost); // Copy from modified vaues to host 

    // Verify values are correct
    try {
      for(int c = 0; c < P2; c++){
        if(host_array[c] != c) {throw "ADD_OP_MATCH";}
      }
    } catch (...) {
      printf("Problem 3 - Elements not added properly!\n");
      hipFree(dev_array);
      free(host_array);
      return 0; 
    }
    printf("Problem 3 - Successfully added i to array[i] with %d elements\n", P2);
    hipFree(dev_array);
    free(host_array); 

    ///////////////////////////////////////////////////////////////////////////////////////////////////////////
    // Problem 4 - same as problem 3, but with 8000 elements 
    byte_size = P4 * sizeof(int);
    dim3 gd(P4);
    hipMalloc((void**) &dev_array, byte_size);  
    initialize<<<1, gd>>>(dev_array);           // Initialize array 
    add<<<1, gd>>>(dev_array);                  // Add index to array           
    host_array = (int*)malloc(byte_size);       
    hipMemcpy(host_array, dev_array, byte_size, hipMemcpyDeviceToHost); 

    // Verify values are correct, 8000 threads might be too much for the device and the answer is incorrect
    try {
      for(int c = 0; c < P4; c++){
        if(host_array[c] != c) {throw "ADD_OP_MATCH";}
      }
    } catch (...) {
      printf("Problem 4 - Elements not added properly!\n");
      hipFree(dev_array);
      free(host_array);
      return 0; 
    }
    printf("Problem 4 - Successfully added i to array[i] with %d elements\n", P4);
    hipFree(dev_array);
    free(host_array);      
    return 0;
}